#include "hip/hip_runtime.h"
/*

Copyright (C) 2018-2020 Deep Tavker (tavkerdeep@gmail.com)

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/


#include <algorithm>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <iostream>
#include <time.h>
#include <vector>
#include <thrust/sort.h>
#include <chrono>

using namespace std;
using namespace std::chrono;




double *x, *y, *z;
double Xmax=9, Xmin=0;
double Ymax=9, Ymin=0;
double Zmax=9, Zmin=0;
double re=0.072, DELTA=0;
int NUM=80000;
int MAX_NEIGHB=1500;
int THREADS_PER_BLOCK=512;

void create_particles(int NUM){
    x = (double *)malloc(sizeof(double)*NUM);
    y = (double *)malloc(sizeof(double)*NUM);
    z = (double *)malloc(sizeof(double)*NUM);


    srand((unsigned)time(0)); 
    double lowest=0, highest=8; 
    double range=(highest-lowest)+1; 
    for(int index=0; index<NUM; index++){ 
        x[index] = lowest+double(range*rand()/(RAND_MAX + 1.0)); 
        y[index] = lowest+double(range*rand()/(RAND_MAX + 1.0)); 
        z[index] = lowest+double(range*rand()/(RAND_MAX + 1.0)); 
    } 
}

int **neighb, **neighb_cuda;
int *particleHash, *particleid, *cellStart, *cellEnd;



// ----------------- CUDA KERNELS -------------------------

__global__ void calcHash(double *d_x, double *d_y, double *d_z, int *d_particleHash,\
	int *d_NUM, double *d_Xmax, double *d_Xmin, double *d_re, double *d_DELTA, double *d_Ymin, \
  double *d_Ymax, double *d_Zmax, double *d_Zmin, int *d_particleid, int *d_tnc, int *ncx, int *ncy,\
  int *ncz){

  int k = threadIdx.x + blockIdx.x * blockDim.x;
  if(k < *d_NUM){


  *ncx = int((*d_Xmax - *d_Xmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in x direction
  *ncy = int((*d_Ymax - *d_Ymin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in y direction
  *ncz = int((*d_Zmax - *d_Zmin) / (*d_re + *d_DELTA)) + 1;     // Number of cells in z direction
  *d_tnc = *ncx * *ncy * *ncz;

  
  int *icell, *jcell, *kcell, *cellNum;

  int sizeint = sizeof(int);
  icell = (int *)malloc(sizeint);
  jcell = (int *)malloc(sizeint);
  kcell = (int *)malloc(sizeint);
  cellNum = (int *)malloc(sizeint);
  
  *icell = int((d_x[k] - *d_Xmin) / (*d_re + *d_DELTA)) + 1;
  *jcell = int((d_y[k] - *d_Ymin) / (*d_re + *d_DELTA)) + 1;
  *kcell = int((d_z[k] - *d_Zmin) / (*d_re + *d_DELTA)) + 1;

  *cellNum = *icell + (*jcell - 1)* *ncx + (*kcell - 1)* *ncx * *ncy;

  d_particleHash[k] = *cellNum;
  d_particleid[k] = k;

  
  free(icell);
  free(jcell);
  free(kcell);
  free(cellNum);
}

}

__global__ void findCellStart(int *particleHash, int *cellStart, int *cellEnd, int *NUM){

  int k = threadIdx.x + blockIdx.x * blockDim.x; // here index value is equal to the cell number which starts with 1 
  if(k < *NUM){
  if (particleHash[k] != particleHash[k+1] && k!= *NUM - 1){
    cellEnd[particleHash[k] - 1] = k;
    cellStart[particleHash[k+1] - 1] = k+1;
  }
  if(k == *NUM - 1){
    cellEnd[particleHash[k] - 1] = k;
  }
    }

  free(&k);            
}

__global__ void createNeighbourArraysCUDA(int *d_neighb, int *cellStart, int *cellEnd, int *particleHash, int *particleid, int *ncx, int *ncy, int *ncz, int *d_max_neighb,  int *d_NUM){

  int index = threadIdx.x + blockIdx.x * blockDim.x; 

  if(index < *d_NUM){
  int pid, icell, jcell, kcell, cellNum;

  cellNum = particleHash[index]; 
  pid = particleid[index];
  
  int neighb_index = pid * (*d_max_neighb + 1);

  kcell = (cellNum - 1)/((*ncx) * (*ncy)) + 1;
  jcell = ((cellNum - 1) - ((kcell - 1)* (*ncx) * (*ncy)))/ *ncx + 1;
  icell = cellNum - 1 - *ncx * (jcell - 1) - (*ncx * *ncy)*(kcell - 1) + 1;

  int Cnum, J;
  int curr_neighb_num = 0;
  
  int row, colu, elev, m1, m2, m3, m4, m5, m6;
  if (icell == 1)m1 = 0; else m1 = -1;
  if (icell == *ncx)m2 = 0; else m2 = +1;
  if (jcell == 1)m3 = 0; else m3 = -1;
  if (jcell == *ncy)m4 = 0; else m4 = +1;
  if (kcell == 1)m5 = 0; else m5 = -1;
  if (kcell == *ncz)m6 = 0; else m6 = +1;

  for (row = m1; row <= m2; row++)
  {
    for (colu = m3; colu <= m4; colu++) 
    {
      for (elev = m5; elev <= m6; elev++)
      {

        Cnum = icell + row + (jcell - 1 + colu)* *ncx + (kcell - 1 + elev)* *ncx* *ncy;

        if (cellEnd[Cnum - 1] != -1){

        for (int JJ = cellStart[Cnum -1]; JJ <= cellEnd[Cnum - 1]; JJ++)
        {
          J = particleid[JJ];
          curr_neighb_num++;
          d_neighb[neighb_index + curr_neighb_num] = J+1; //here the index is shifted by one unit to conform to the original MPS convention
          
        }
      }
      }
    }
  }
  
  
  d_neighb[neighb_index] = curr_neighb_num;
 }
}

__global__ void InitializeCellDetails(int *cellStart, int *cellEnd, int *d_tnc){
  int index = threadIdx.x + blockIdx.x * blockDim.x; 
  if(index < *d_tnc){
  cellStart[index] = 0; cellEnd[index] = -1;
}
free(&index);
}

__global__ void Template(int *particleHash, int *particleid, int *cellStart, int *cellEnd, int *ncx, int *ncy, int *ncz, int *size_neighbours, int *test){
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int pid, icell, jcell, kcell, cellNum;
  int *neighbours;
  neighbours = (int *)malloc(*size_neighbours);
  cellNum = particleHash[index]; 
  pid = particleid[index];

  kcell = (cellNum - 1)/((*ncx) * (*ncy)) + 1;
  jcell = ((cellNum - 1) - ((kcell - 1)* (*ncx) * (*ncy)))/ *ncx + 1;
  icell = cellNum - 1 - *ncx * (jcell - 1) - (*ncx * *ncy)*(kcell - 1) + 1;

  int Cnum, J;
  int curr_neighb_num = 0;
  
  int row, colu, elev, m1, m2, m3, m4, m5, m6;
  if (icell == 1)m1 = 0; else m1 = -1;
  if (icell == *ncx)m2 = 0; else m2 = +1;
  if (jcell == 1)m3 = 0; else m3 = -1;
  if (jcell == *ncy)m4 = 0; else m4 = +1;
  if (kcell == 1)m5 = 0; else m5 = -1;
  if (kcell == *ncz)m6 = 0; else m6 = +1;

  for (row = m1; row <= m2; row++)
  {
    for (colu = m3; colu <= m4; colu++) 
    {
      for (elev = m5; elev <= m6; elev++)
      {

        Cnum = icell + row + (jcell - 1 + colu)* *ncx + (kcell - 1 + elev)* *ncx* *ncy;

        if (cellEnd[Cnum - 1] != -1){

        for (int JJ = cellStart[Cnum -1]; JJ <= cellEnd[Cnum - 1]; JJ++)
        {
          J = particleid[JJ];
          curr_neighb_num++;
          neighbours[curr_neighb_num] = J;
          
        }
      }
      }
    }
  }
  
  
  neighbours[0] = curr_neighb_num;
  test[pid] = curr_neighb_num;

  //any further operations can be done using this neighbour array

}




// ------------------------- Host sub-sub-routine for neighbour computation ------------------------ 

void neighbour_cuda_1(){

  //cout<<endl<<"Time study for neighbour_cuda_1()"<<endl;

  // ------------------ variable declarations and initializations ------------------------------

  int *d_cellEnd, *d_cellStart, *d_NUM, *d_tnc, *tnc, *d_ncx, *d_ncy, *d_ncz, *d_max_neighb;
  int *d_particleHash, *d_particleid, *d_neighb, *h_neighb, *d_sizeof_neighbours;
  double *d_x, *d_y, *d_z, *d_Xmax, *d_Xmin, *d_Ymax, *d_Ymin, *d_Zmax, *d_Zmin, *d_re, *d_DELTA;

  int arrsizeint = NUM * sizeof(int);
  int sizeint = sizeof(int);
  int arrsizedouble = NUM * sizeof(double);
  int sizedouble = sizeof(double);
  int sizeneighb = NUM * (MAX_NEIGHB + 1) * sizeof(int);
  int sizeof_neighbours = (MAX_NEIGHB + 1) * sizeof(int);

  tnc = (int *)malloc(sizeint);
  h_neighb = (int *)malloc(sizeneighb);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  hipMalloc((void **)&d_particleHash, arrsizeint);
  hipMalloc((void **)&d_particleid, arrsizeint); 
  hipMalloc((void **)&d_x, arrsizedouble);
  hipMalloc((void **)&d_y, arrsizedouble);
  hipMalloc((void **)&d_z, arrsizedouble);
  hipMalloc((void **)&d_Xmin, sizedouble);
  hipMalloc((void **)&d_Xmax, sizedouble);
  hipMalloc((void **)&d_Ymin, sizedouble);
  hipMalloc((void **)&d_Ymax, sizedouble);
  hipMalloc((void **)&d_Zmin, sizedouble);
  hipMalloc((void **)&d_Zmax, sizedouble);
  hipMalloc((void **)&d_re, sizedouble);
  hipMalloc((void **)&d_DELTA, sizedouble);
  hipMalloc((void **)&d_NUM, sizeint);
  hipMalloc((void **)&d_tnc, sizeint);
  hipMalloc((void **)&d_ncx, sizeint);
  hipMalloc((void **)&d_ncy, sizeint);
  hipMalloc((void **)&d_ncz, sizeint);
  hipMalloc((void **)&d_neighb, sizeneighb);
  hipMalloc((void **)&d_max_neighb, sizeint);
  hipMalloc((void **)&d_sizeof_neighbours, sizeof_neighbours);

  hipMemcpy(d_x, x, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmin, &Xmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmax, &Xmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymin, &Ymin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymax, &Ymax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmin, &Zmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmax, &Zmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_re, &re, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_DELTA, &DELTA, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_NUM, &NUM, sizeint, hipMemcpyHostToDevice);
  hipMemcpy(d_max_neighb, &MAX_NEIGHB, sizeint, hipMemcpyHostToDevice);
  hipMemcpy(d_sizeof_neighbours, &sizeof_neighbours, sizeint, hipMemcpyHostToDevice);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  //cout<<" initial memory transfers and allocations : "<<milliseconds<<endl;

  

  // --------------- running the calcHash kernel ----------------------------------------
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  calcHash<<<NUM/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_particleHash, d_NUM, d_Xmax, d_Xmin, d_re, d_DELTA, d_Ymin, d_Ymax, d_Zmax, d_Zmin, d_particleid, d_tnc, d_ncx, d_ncy, d_ncz);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  //cout<<" calcHash : "<<milliseconds<<endl;
  // ---------------- sorting the particleHash array -----------------------------

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::device_ptr<int> dev_Hash(d_particleHash);
  thrust::device_ptr<int> dev_id(d_particleid);
  thrust::sort_by_key(dev_Hash, dev_Hash + NUM, dev_id); //need to generalise this 10
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  //cout<<" Radix-Sort : "<<milliseconds<<endl;
  
  // --------------------- finding cell start and cell end for each cell -----------------------------

  hipMemcpy(tnc, d_tnc, sizeint, hipMemcpyDeviceToHost);
  int cellarrsize = *tnc * sizeof(int);
  cellStart = (int *)malloc(cellarrsize);
  cellEnd = (int *)malloc(cellarrsize);
  hipMalloc((void **)&d_cellStart, cellarrsize); 
  hipMalloc((void **)&d_cellEnd, cellarrsize); 

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  InitializeCellDetails<<<*tnc/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_cellStart, d_cellEnd, d_tnc);
  findCellStart<<<NUM/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_particleHash, d_cellStart, d_cellEnd, d_NUM);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  //cout<<" InitializeCellDetails and findCellStart : "<<milliseconds<<endl;
  
  // -------------------------- Creating neighbour arrays for each particle ------------------------------

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  createNeighbourArraysCUDA<<<NUM/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_neighb, d_cellStart, d_cellEnd, d_particleHash, d_particleid, d_ncx, d_ncy, d_ncz, d_max_neighb, d_NUM);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

 //cout<<" createNeighbourArraysCUDA : "<<milliseconds<<endl;
  
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  hipMemcpy(h_neighb, d_neighb, sizeneighb, hipMemcpyDeviceToHost);



  // ---------------------------- Populating neighb array ----------------------
  
       
  neighb_cuda = new int*[NUM+1];
  for(int i=0; i<NUM+1; i++){
    neighb_cuda[i] = new int[MAX_NEIGHB + 2];
  }
  
  //neighb_cuda[10][50] = 5;
  
  
  for(int j=0; j<NUM; j++){
    for(int i=0; i<h_neighb[j*(MAX_NEIGHB + 1)]; i++){
      neighb_cuda[j+1][i+2] = h_neighb[j*(MAX_NEIGHB + 1) + i + 1];
    }
    neighb_cuda[j+1][1] = h_neighb[j*(MAX_NEIGHB + 1)];
  }
  
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  //cout<<" neighbour array transfer and construction of 2D neighb : "<<milliseconds<<endl;
  
  
  // -------------------------- Deallocating memory ---------------------------

  hipFree(d_particleHash);
  hipFree(d_particleid);
  hipFree(d_cellStart);
  hipFree(d_cellEnd);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_Xmin);
  hipFree(d_Xmax);
  hipFree(d_Ymin);
  hipFree(d_Ymax);
  hipFree(d_Zmin);
  hipFree(d_Zmax);
  hipFree(d_re);
  hipFree(d_NUM);
  hipFree(d_tnc);
  hipFree(d_ncx);
  hipFree(d_ncy);
  hipFree(d_ncz);
  hipFree(d_neighb);
  hipFree(d_max_neighb);
  hipFree(d_sizeof_neighbours);

  free(h_neighb);
  free(tnc);
}

void NEIGHBOUR_serial(){

 // cout<<endl<<"Time study for NEIGHBOUR_serial()"<<endl;

  // ------------------PARAMETERS DEFENTION -------------------------------------
  int ncx = int((Xmax - Xmin) / (re + DELTA)) + 1;     // Number of cells in x direction
  int ncy = int((Ymax - Ymin) / (re + DELTA)) + 1;     // Number of cells in y direction
  int ncz = int((Zmax - Zmin) / (re + DELTA)) + 1;     // Number of cells in z direction

  int tnc = ncx*ncy*ncz;                 // Total number of cells   
  int m, k, kmax, Cnum;

  neighb = new int*[NUM+1];
  for(int i=0; i<NUM+1; i++){
    neighb[i] = new int[MAX_NEIGHB + 2];
  }


  int *Ista, *Iend, *nc, *icell, *jcell, *kcell;
  int *ip;                             // I is sorted number of ip[I] th paricle
  Ista = new int[tnc + 1]; //this points to the index of the first element in a cell in the array ip
  Iend = new int[tnc + 1]; //index of the last element in a cell in the array ip
  nc = new int[tnc + 1];
  icell = new int[NUM + 1];
  jcell = new int[NUM + 1];
  kcell = new int[NUM + 1];
  ip = new int[NUM + 1]; //this is the main array that we are looking for, it is sorted 
  // according to cell numbers and it contains particle indices 



  //----------------- ALLOCATING PRTICLES IN CELLS --------------------------


  for (k = 1; k <= tnc; k++) //cell loop 
  {
    Ista[k] = 1;
    Iend[k] = 0;
    nc[k] = 0;
  }
  for (k = 1; k <= NUM; k++) //particle loop
  {
    icell[k] = int((x[k-1] - Xmin) / (re + DELTA)) + 1;
    jcell[k] = int((y[k-1] - Ymin) / (re + DELTA)) + 1;
    kcell[k] = int((z[k-1] - Zmin) / (re + DELTA)) + 1;

    Cnum = icell[k] + (jcell[k] - 1)*ncx + (kcell[k] - 1)*ncx*ncy;     // Cell number in which particle k located

    nc[Cnum]++;                       // Number of particle in cell Cnum
    Iend[Cnum]++;                   // Number of particle in cell Cnum 

    for (m = Iend[tnc]; m >= Iend[Cnum]; m--)
    {
      if (m>0) ip[m + 1] = ip[m];
    } //this block is there to create space at the end as and when new particles are added

    for (m = Cnum + 1; m <= tnc; m++)
    {
      Ista[m]++;
      Iend[m]++;
    }

    ip[Iend[Cnum]] = k;
  }



  //--------------- FINDIND NEIGHBORS ----------------------------------
  int JJ, J;
  for (int I = 1; I <= NUM; I++)
  {
    k = 2;
    int row, colu, elev, m1, m2, m3, m4, m5, m6;
    if (icell[I] == 1)m1 = 0; else m1 = -1;
    if (icell[I] == ncx)m2 = 0; else m2 = +1;
    if (jcell[I] == 1)m3 = 0; else m3 = -1;
    if (jcell[I] == ncy)m4 = 0; else m4 = +1;
    if (kcell[I] == 1)m5 = 0; else m5 = -1;
    if (kcell[I] == ncz)m6 = 0; else m6 = +1;

    for (row = m1; row <= m2; row++) //could be -1 to 1 , the triple loop is basically there to find all the 9 cells around that particle, including the one in which it itself is
    {
      for (colu = m3; colu <= m4; colu++) 
      {
        for (elev = m5; elev <= m6; elev++)
        {

          Cnum = icell[I] + row + (jcell[I] - 1 + colu)*ncx + (kcell[I] - 1 + elev)*ncx*ncy;

          for (JJ = Ista[Cnum]; JJ <= Iend[Cnum]; JJ++)
          {
            J = ip[JJ]; //J is tha ACTUAL particle index 
            neighb[I][k] = J;
            k++;
          }
        }
      }
    }
    kmax = k - 2;
    neighb[I][1] = kmax; //this is the total number of neighbours, which is stored at the beginning 
    //if( neighb[I][1]>1098 ||neighb[I][1]*0!=0) printf("ERROR, the neighbors of particles %d is %d", I, neighb[I][1]);
  }
  //--------------------Clearing dynamic arrays ----------------------------

  delete[]Ista;
  delete[]Iend;
  delete[]nc;
  delete[]icell;
  delete[]jcell;
  delete[]kcell;
  delete[]ip;
  Ista = NULL; Iend = NULL; nc = NULL; icell = NULL; jcell = NULL; kcell = NULL, ip = NULL;
}

void neighbour_cuda_2(){

 // cout<<endl<<"Time study for neighbour_cuda_2()"<<endl;

  // ------------------ variable declarations and initializations ------------------------------

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  int *d_cellEnd, *d_cellStart, *d_NUM, *d_tnc, *tnc, *d_ncx, *d_ncy, *d_ncz;
  int *d_particleHash, *d_particleid;
  double *d_x, *d_y, *d_z, *d_Xmax, *d_Xmin, *d_Ymax, *d_Ymin, *d_Zmax, *d_Zmin, *d_re, *d_DELTA;

  int arrsizeint = NUM * sizeof(int);
  int sizeint = sizeof(int);
  int arrsizedouble = NUM * sizeof(double);
  int sizedouble = sizeof(double);
  
  particleHash = (int *)malloc(arrsizeint);
  particleid = (int *)malloc(arrsizeint);
  tnc = (int *)malloc(sizeint);

  hipMalloc((void **)&d_particleHash, arrsizeint);
  hipMalloc((void **)&d_particleid, arrsizeint); 
  
  hipMalloc((void **)&d_x, arrsizedouble);
  hipMalloc((void **)&d_y, arrsizedouble);
  hipMalloc((void **)&d_z, arrsizedouble);
  hipMalloc((void **)&d_Xmin, sizedouble);
  hipMalloc((void **)&d_Xmax, sizedouble);
  hipMalloc((void **)&d_Ymin, sizedouble);
  hipMalloc((void **)&d_Ymax, sizedouble);
  hipMalloc((void **)&d_Zmin, sizedouble);
  hipMalloc((void **)&d_Zmax, sizedouble);
  hipMalloc((void **)&d_re, sizedouble);
  hipMalloc((void **)&d_DELTA, sizedouble);
  hipMalloc((void **)&d_NUM, sizeint);
  hipMalloc((void **)&d_tnc, sizeint);
  hipMalloc((void **)&d_ncx, sizeint);
  hipMalloc((void **)&d_ncy, sizeint);
  hipMalloc((void **)&d_ncz, sizeint);

  hipMemcpy(d_x, x, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, arrsizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmin, &Xmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Xmax, &Xmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymin, &Ymin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Ymax, &Ymax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmin, &Zmin, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_Zmax, &Zmax, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_re, &re, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_DELTA, &DELTA, sizedouble, hipMemcpyHostToDevice);
  hipMemcpy(d_NUM, &NUM, sizeint, hipMemcpyHostToDevice);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  cout<<" memory initializations and allocations : "<<milliseconds<<endl;

  // --------------- running the calcHash kernel ----------------------------------------

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  calcHash<<<NUM/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_particleHash, d_NUM, d_Xmax, d_Xmin, d_re, d_DELTA, d_Ymin, d_Ymax, d_Zmax, d_Zmin, d_particleid, d_tnc, d_ncx, d_ncy, d_ncz);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  cout<<" calcHash : "<<milliseconds<<endl;
  // ---------------- sorting the particleHash array -----------------------------

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  thrust::device_ptr<int> dev_Hash(d_particleHash);
  thrust::device_ptr<int> dev_id(d_particleid);
  thrust::sort_by_key(dev_Hash, dev_Hash + NUM, dev_id); //need to generalise this 10
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  cout<<" Radix-Sort : "<<milliseconds<<endl;
  
  // --------------------- finding cell start and cell end for each cell -----------------------------

  hipMemcpy(tnc, d_tnc, sizeint, hipMemcpyDeviceToHost);
  int cellarrsize = *tnc * sizeof(int);
  hipMalloc((void **)&d_cellStart, cellarrsize); 
  hipMalloc((void **)&d_cellEnd, cellarrsize); 

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  InitializeCellDetails<<<*tnc/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_cellStart, d_cellEnd, d_tnc);
  findCellStart<<<NUM/THREADS_PER_BLOCK + 1,THREADS_PER_BLOCK>>>(d_particleHash, d_cellStart, d_cellEnd, d_NUM);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  cout<<" InitializeCellDetails and findCellStart : "<<milliseconds<<endl;

  // ------------------- Transferring the required arrays into global memory -----

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  hipMemcpy(particleHash, d_particleHash, arrsizeint, hipMemcpyDeviceToHost);
  hipMemcpy(particleid, d_particleid, arrsizeint, hipMemcpyDeviceToHost);
  hipMemcpy(cellStart, d_cellStart, cellarrsize, hipMemcpyDeviceToHost);
  hipMemcpy(cellEnd, d_cellEnd, cellarrsize, hipMemcpyDeviceToHost);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  cout<<" memory transfers of 4 main arrays : "<<milliseconds<<endl;

  
  cout<<endl;
  

  // -------------------------- Deallocating memory ---------------------------

  hipFree(d_particleHash);
  hipFree(d_particleid);
  hipFree(d_cellStart);
  hipFree(d_cellEnd);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_Xmin);
  hipFree(d_Xmax);
  hipFree(d_Ymin);
  hipFree(d_Ymax);
  hipFree(d_Zmin);
  hipFree(d_Zmax);
  hipFree(d_re);
  hipFree(d_NUM);
  hipFree(d_tnc);
  hipFree(d_ncx);
  hipFree(d_ncy);
  hipFree(d_ncz);

  free(particleHash);
  free(particleid);
  free(cellStart);
  free(cellEnd);
  free(tnc);
}





// -------------------- host sub-routine for neighbour calculation --------------------------

int main(){
  for(int k=1; k<10; k++){
    NUM= k*10000;
    for(int m=1; m<11; m++){


    THREADS_PER_BLOCK = pow(2, m);
    create_particles(NUM);
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
  	neighbour_cuda_1();
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    //NEIGHBOUR_serial();
    //high_resolution_clock::time_point t3 = high_resolution_clock::now();
    //neighbour_cuda_2();
    //high_resolution_clock::time_point t4 = high_resolution_clock::now();
   // bool test_num = true, test_id = true;
   // for(int i=0; i<NUM; i++){
   //   for(int j=0; j<neighb[i+1][1]; j++){
   //     test_id = test_id * (neighb[i+1][j+2] == neighb_cuda[i+1][j+2]);
    //  }
    //  test_num = test_num *  (neighb[i+1][1] == neighb_cuda[i+1][1]);
   // }

   // if(test_num == true && test_id == true){
   // cout<<"All tests passing"<<endl;
  // }

    auto duration1 = duration_cast<milliseconds>( t2 - t1 ).count();
   // auto duration2 = duration_cast<milliseconds>( t3 - t2 ).count();
    //auto duration3 = duration_cast<milliseconds>( t4 - t3 ).count();
    cout <<NUM<<","<<THREADS_PER_BLOCK<<","<<duration1<<endl;
    }
  }

	return 0;
} 
